#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define BLOCK_SIZE 256

// Kernel function to generate and print random numbers
__global__ void randomKernel(unsigned int seed, int count) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialize random number generator
    hiprandStateXORWOW_t state;
    hiprand_init(seed, tid, 0, &state);
    int g = 0;
    // Generate and print random numbers
    for (int x = 0; x < 40000; x++) {
        for (int i = tid; i < count; i += blockDim.x * gridDim.x) {
            float randNum = hiprand_uniform(&state);
            //printf("%d, %f\n",i*x, randNum);
        }
        
    }
}

int main() {
    int count = 1000000;
    int blockSize = BLOCK_SIZE;
    int gridSize = (count + blockSize - 1) / blockSize;

    // Generate and print random numbers on the device
    randomKernel << <gridSize, blockSize >> > (time(NULL), count);

    // Ensure all the print statements have completed
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        return -1;
    }
    return 0;
}
